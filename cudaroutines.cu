#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "book.h"
#include "cuda_bridge.h"

#define BLOCK_SIZE 16

__global__ void gpu_square_matrix_mult(int *d_a, int *d_b, int *d_result, int n) 
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    #pragma unroll
    for (int sub = 0; sub < gridDim.x; ++sub) 
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if(idx >= n*n)
        {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if(idx >= n*n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }  
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < n && col < n)
    {
        d_result[row * n + col] = tmp % 512;
    }
}
int *Md = NULL, *Nd = NULL, *Pd = NULL;

void MatrixMultiplication(int *&M, int *&N, int *&P, int Width) {

  int size = Width * Width * sizeof(int);

  // allocate memory on the GPU
  //if(Md == NULL) {
    //std::cout << "hipMalloc" << std::endl;
    HANDLE_ERROR( hipMalloc((void**)&Md, size) );
  //} else {
    //std::cout << "ALLOCATED" << std::endl;
  //}
  //if(Nd == NULL)
    HANDLE_ERROR( hipMalloc((void**)&Nd, size) );
  //if(Pd == NULL)
    HANDLE_ERROR( hipMalloc((void**)&Pd, size) );

  // transfer M and N to device memory
  HANDLE_ERROR( hipMemcpy(Md, M, size, hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(Nd, N, size, hipMemcpyHostToDevice) );

  unsigned int grid_rows = (Width + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int grid_cols = (Width + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 dimGrid(grid_cols, grid_rows);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  //std::cout << "Width: " << Width << std::endl;
  //std::cout << "Width/32: " << Width/32 << std::endl;

  //Kernel<<<dimGrid, dimBlock>>>( Md, Nd, Pd, Width);
  gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(Md, Nd, Pd, Width);
  //std::cout << "Width/32: " << Width/32 << std::endl;

  // transfer P from device     
  //std::cout << "P: " << P[0] << std::endl;
  HANDLE_ERROR( hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost) );
  hipDeviceSynchronize();

  HANDLE_ERROR( hipFree(Md) );
  HANDLE_ERROR( hipFree(Nd) );
  HANDLE_ERROR( hipFree(Pd) );
}

